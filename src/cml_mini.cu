#include "hip/hip_runtime.h"
#include <time.h>
#include <sys/time.h>
//#include <array>>
#include <malloc.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <unistd.h>
#include <math.h>
#include <cblas.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#define L 140
#define L_power 19600
/*
 *this program is used to test cuda-function
 * 1.can cublas in kernel read data correctly?
 * 2.can cublas in kernel compute correctly?
 * 3.the data returened by kernel is correctly?
 *
 *
 *
 *
 */


__global__ void mykernel(float *d_data,float *d_result){
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    hipblasHandle_t handle;
    hipblasStatus_t status;
    status=hipblasCreate(&handle);
    const float alpha=1.0;
    const float beta=1.0;
    status=hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,L,L,L,&alpha,&d_data[L_power*id],L,&d_data[L_power*id],L,&beta,&d_result[L_power*id],L);
    hipblasDestroy(handle);

}

__global__ void simplekernel(float *d_process,float *d_max_value){
    float maxvalue;
    int i;
    maxvalue=d_process[0];
    for (i=0;i<L_power;i++){
        if (maxvalue>d_process[i]){
            maxvalue=d_process[i];
        }
    }
    *d_max_value=maxvalue;
}


/*
__global__  void nocublas_kernel(float *d_data,float *d_result){
	int id = threadIdx.x+blockDim.x*blockIdx.x;
	int i,j,k,m;
	for (i=0;i<L;i++){
		for (j=0;j<L;j++){
			d_result[id*L_power+i*L+j]=0.0;
			for (k=0;k<L;k++){
				for (m=0;m<L;m++){
					d_result[id*L_power+i*L+j]+=d_data[L_power*id+i*L+k]*d_data[L_power*id+m*L+j];
					}
				}
			}
		}

}
*/

int main(int argc,char *argv[]){
    int oc;
    FILE *fdata;
    char *datafilename;
    while((oc = getopt(argc, argv,"f:")) !=-1){
        switch(oc)
        {
        case 'f':
            datafilename=optarg;
            break;
        }
    }
//    datafilename="~/..";
    fdata=fopen(datafilename,"rb");
    float *matrix;
    const float alpha=1.0;
    const float beta=0.0;
    int N=50;
    matrix = new float [L_power*N];
    fseek(fdata,0,SEEK_SET);
    fread(matrix,sizeof(float),N*L_power,fdata);

    float *result;
    result = new float [L_power*N];

    float *d_data;
//    float *d_result;
    float *d_process[N];
    float *max_value[N];
    float *d_max_value[N];

    hipMalloc((void **) &d_data,sizeof(float)*N*L_power);
//    hipMalloc((void **) &d_result,sizeof(float)*N*L_power);
    hipMalloc((void **) &d_max_value,sizeof(float)*N);

    hipMemcpy(d_data,matrix,sizeof(float)*N*L_power,hipMemcpyHostToDevice);
//    mykernel<<<1,10>>>(d_data,d_result);
    //    nocublas_kernel<<<1,10>>>(d_data,d_result);
//    hipDeviceSynchronize();
    //hipMemcpy(result,d_result,sizeof(float)*N*L_power,hipMemcpyDeviceToHost);

    hipStream_t stream[N];
    //hipblasHandle_t handle[N];
    for(int i=0;i<N;i++){
        hipStreamCreate(&stream[i]);
        hipblasCreate(&handle[i]);
    }
    for(int i=0;i<N;i++){
        hipblasSetStream(handle[i],stream[i]);
    }
    for(int i=0;i<N;i++){
        float *d_temp;
        hipMalloc((void **)&d_temp,sizeof(float)*L_power);
        hipblasSgemm(handle[i],HIPBLAS_OP_T,HIPBLAS_OP_N,L,L,L,&alpha,&d_data[L_power*i],L,&d_data[L_power*i],L,&beta,&d_temp,L);
        simplekernel<<<1,1,stream[i]>>>(&d_temp,&d_max_value[i]);
        hipFree(d_temp);
    //mykernel<<<1,1,0,stream[i]>>>(&d_data[L_power*i],&d_result[L_power*i]);
    }
    hipDeviceSynchronize();
    hipMemcpy(result,d_result,sizeof(float)*N*L_power,hipMemcpyDeviceToHost);
    hipMemcpy(max_value,d_max_value,sizeof(float)*N,hipMemcpyDeviceToHost);
    for (int i=0;i<N;i++){
	hipblasDestroy(handle[i]);
	}
    float *Host_result;
    float *Host_max[N];
    Host_result = new float [N*L_power];
    for (int i=0;i<N;i++){
        cblas_sgemm(CblasRowMajor,CblasNoTrans,CblasTrans,L,L,L,1,&matrix[i*L_power],L,&matrix[i*L_power],L,0,&Host_result[i*L_power],L);
    }
    for (int i=0;i<N;i++){
        Host_max[i]=Host_result[i*L_power];
        for (int j=0;j<L_power;j++){
                if (Host_max[i]>Host_result[i*L_power+j]){
                    Host_max[i]=Host_result[i*L_power+j];
                }
        }
    }
    /*
    for (int i=0;i<N;i++){
        float diff=0.0f;
        for (int j=0;j<L_power;j++){
            diff+=(Host_result[i*L_power+j]-result[i*L_power+j])*(Host_result[i*L_power+j]-result[i*L_power+j]);
        }
        printf("diff %d\t%f\n",i,sqrt(diff/(L_power*L_power)));
    }
    */
    for (int i=0;i<N;i++){
        printf("%d\t%f\t%f\n",i,max_value[i],Host_max[i]);
    }

    hipFree(d_data);
    hipFree(d_result);
    delete[] matrix;
    delete[] result;
    delete[] Host_result;
    fclose(fdata);
    return 1;

}
